#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

// You should modify these parameters.
#define BITWIDTH 8
// #define STRIDE 1

constexpr int times = 4096;

__global__ void test_shmem(int STRIDE) {
#if (BITWIDTH == 2)
    volatile __shared__ uint16_t shm[32 * 128];
    volatile uint16_t tmp;
#elif (BITWIDTH == 4)
    volatile __shared__ uint32_t shm[32 * 128];
    volatile uint32_t tmp;
#elif (BITWIDTH == 8)
    volatile __shared__ uint64_t shm[32 * 128];
    volatile uint64_t tmp;
#endif

    for (int i = 0; i < times; i++) {
        tmp = shm[threadIdx.x * STRIDE];
    }
}

int main() {
    for (int STRIDE = 1; STRIDE <= 32; STRIDE *= 2){
        int size = (1 << 16);
        dim3 gridSize(size / 128, 1);
        dim3 blockSize(128, 1);

        hipEvent_t st, ed;
        hipEventCreate(&st);
        hipEventCreate(&ed);
        float duration;

        // Warm up.
        for (int t = 0; t < 1024; t++) {
            test_shmem<<<gridSize, blockSize>>>(STRIDE);
            cudaCheckError();
        }

        hipEventRecord(st, 0);
        for (int t = 0; t < 1024; t++) {
            test_shmem<<<gridSize, blockSize>>>(STRIDE);
            cudaCheckError();
        }
        hipEventRecord(ed, 0);
        hipEventSynchronize(st);
        hipEventSynchronize(ed);
        hipEventElapsedTime(&duration, st, ed);
        duration /= float(1024) * float(times);
        
        std::cout << "bitwidth:  " << BITWIDTH << std::endl;
        std::cout << "stride:    " << STRIDE << std::endl;
        std::cout << "bandwidth: " << size * BITWIDTH / duration / 1e6 << std::endl;
    }
    
}

