
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <cmath>
#include <sstream>
#include <cassert>
#include <fstream>

const int H = 10000, W = 10000;
const int kernel_size = 3;
const int n = (W + kernel_size - 1) * (H + kernel_size - 1);
float *input, *output;
float *dev_input, *dev_output;
float *answer = nullptr;
float *produce = nullptr;

std::ofstream oFile;

#define checkCudaErrors(status)                   \
    do                                            \
    {                                             \
        if (status != 0)                          \
        {                                         \
            fprintf(stderr, "CUDA failure at [%s] (%s:%d): %s\n", __PRETTY_FUNCTION__, __FILE__, __LINE__, hipGetErrorString(status)); \
            hipDeviceReset();                    \
            abort();                              \
        }                                         \
    } while (0)

void prepareInput()
{
    input = new float[n];
    output = new float[n];
    for (int i = 0; i < n; i++)
    {
        // input[i] = 1.f;
        input[i] = (((size_t)i * 373587883 + 472882027) % 2147483647) / 2147483647.f;
    }

    checkCudaErrors(hipMalloc(&dev_input, n * sizeof(float)));
    checkCudaErrors(hipMalloc(&dev_output, H * W * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_input, input, n * sizeof(float), hipMemcpyHostToDevice));
}

__device__ inline float calc(float input)
{
    // return input;
    return sqrtf(expf(input / 4.0 + 0.24));
}

__global__ void naive(float *dev_input, float *dev_output, int H, int W, int kernel_size)
{
    int thread_id_x = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (thread_id_x >= W || thread_id_y >= H)
        return;
    float ans = 0.f;
    int init_pos = thread_id_y * (W + kernel_size - 1) + thread_id_x;
    ans += calc(dev_input[init_pos]);
    ans += calc(dev_input[init_pos + 1]);
    ans += calc(dev_input[init_pos + 2]);
    ans += calc(dev_input[init_pos + W + kernel_size - 1]);
    ans += calc(dev_input[init_pos + W + kernel_size]);
    ans += calc(dev_input[init_pos + W + kernel_size + 1]);
    ans += calc(dev_input[init_pos + 2 * (W + kernel_size - 1)]);
    ans += calc(dev_input[init_pos + 2 * (W + kernel_size - 1) + 1]);
    ans += calc(dev_input[init_pos + 2 * (W + kernel_size - 1) + 2]);
    dev_output[thread_id_y * W + thread_id_x] = ans;
}

__global__ void shared_mem(float *dev_input, float *dev_output, int H, int W, int kernel_size)
{
    int thread_id_x = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (thread_id_x >= W || thread_id_y >= H)
        return;
    extern __shared__ float cache[];
    int init_pos = thread_id_y * (W + kernel_size - 1) + thread_id_x;
    cache[threadIdx.y * (blockDim.x + kernel_size - 1) + threadIdx.x] = calc(dev_input[init_pos]);
    bool x_last = threadIdx.x == blockDim.x - 1 || thread_id_x == W - 1;
    bool y_last = threadIdx.y == blockDim.y - 1 || thread_id_y == H - 1;
    if (x_last)
    {
        cache[threadIdx.y * (blockDim.x + kernel_size - 1) + threadIdx.x + 1] = calc(dev_input[init_pos + 1]);
        cache[threadIdx.y * (blockDim.x + kernel_size - 1) + threadIdx.x + 2] = calc(dev_input[init_pos + 2]);
    }
    if (y_last)
    {
        cache[(threadIdx.y + 1) * (blockDim.x + kernel_size - 1) + threadIdx.x] = calc(dev_input[init_pos + (W + kernel_size - 1)]);
        cache[(threadIdx.y + 2) * (blockDim.x + kernel_size - 1) + threadIdx.x] = calc(dev_input[init_pos + 2 * (W + kernel_size - 1)]);
    }
    if (x_last && y_last)
    {
        cache[(threadIdx.y + 1) * (blockDim.x + kernel_size - 1) + threadIdx.x + 1] = calc(dev_input[init_pos + W + kernel_size - 1 + 1]);
        cache[(threadIdx.y + 1) * (blockDim.x + kernel_size - 1) + threadIdx.x + 2] = calc(dev_input[init_pos + W + kernel_size - 1 + 2]);
        cache[(threadIdx.y + 2) * (blockDim.x + kernel_size - 1) + threadIdx.x + 1] = calc(dev_input[init_pos + 2 * (W + kernel_size - 1) + 1]);
        cache[(threadIdx.y + 2) * (blockDim.x + kernel_size - 1) + threadIdx.x + 2] = calc(dev_input[init_pos + 2 * (W + kernel_size - 1) + 2]);
    }
    __syncthreads();
    float ans = 0.f;
    ans += cache[threadIdx.y * (blockDim.x + kernel_size - 1) + threadIdx.x];
    ans += cache[threadIdx.y * (blockDim.x + kernel_size - 1) + threadIdx.x + 1];
    ans += cache[threadIdx.y * (blockDim.x + kernel_size - 1) + threadIdx.x + 2];
    ans += cache[(threadIdx.y + 1) * (blockDim.x + kernel_size - 1) + threadIdx.x];
    ans += cache[(threadIdx.y + 1) * (blockDim.x + kernel_size - 1) + threadIdx.x + 1];
    ans += cache[(threadIdx.y + 1) * (blockDim.x + kernel_size - 1) + threadIdx.x + 2];
    ans += cache[(threadIdx.y + 2) * (blockDim.x + kernel_size - 1) + threadIdx.x];
    ans += cache[(threadIdx.y + 2) * (blockDim.x + kernel_size - 1) + threadIdx.x + 1];
    ans += cache[(threadIdx.y + 2) * (blockDim.x + kernel_size - 1) + threadIdx.x + 2];
    dev_output[thread_id_y * W + thread_id_x] = ans;
}

void validate(float *a1, float *a2)
{
    for (int i = 0; i < H * W; ++i)
    {
        if (abs(a1[i] - a2[i]) > 1e-5)
        {
            std::cout << "error answer: " << i << ' ' << a1[i] << ' ' << a2[i] << std::endl;
            exit(1);
        }
    }
}

void test_func(int block_size_x, int block_size_y, bool use_shared_memory = 0, bool validation = 0)
{
    std::string mode = "naive";
    if (use_shared_memory)
        mode = "shared_memory";
    namespace ch = std::chrono;
    double t = 0;
    int test = 20;
    int warmup = 10;
    dim3 grid, block;
    block.x = block_size_x;
    block.y = block_size_y;
    grid.x = (W + block.x - 1) / block.x;
    grid.y = (H + block.y - 1) / block.y;

    if (validation)
    {
        if (answer == nullptr)
        {
            answer = new float[H * W];
            produce = new float[H * W];
            naive<<<grid, block>>>(dev_input, dev_output, H, W, kernel_size);
            checkCudaErrors(hipMemcpy(answer, dev_output, H * W * sizeof(float), hipMemcpyDefault));
        }
        if (use_shared_memory)
        {
            shared_mem<<<grid, block, (block_size_x + kernel_size - 1) * (block_size_y + kernel_size - 1) * sizeof(float)>>>(dev_input, dev_output, H, W, kernel_size);
        }
        else
        {
            naive<<<grid, block>>>(dev_input, dev_output, H, W, kernel_size);
        }
        checkCudaErrors(hipMemcpy(produce, dev_output, H * W * sizeof(float), hipMemcpyDefault));
        validate(produce, answer);
    }

    for (int i = 0; i < warmup + test; i++)
    {
        checkCudaErrors(hipDeviceSynchronize());
        auto beg = ch::high_resolution_clock::now();
        if (use_shared_memory)
        {
            shared_mem<<<grid, block, (block_size_x + kernel_size - 1) * (block_size_y + kernel_size - 1) * sizeof(float)>>>(dev_input, dev_output, H, W, kernel_size);
        }
        else
        {
            naive<<<grid, block>>>(dev_input, dev_output, H, W, kernel_size);
        }
        checkCudaErrors(hipDeviceSynchronize());
        auto end = ch::high_resolution_clock::now();
        double dur = ch::duration_cast<ch::duration<double>>(end - beg).count() * 1000; // ms
        if (i >= warmup)
            t += dur;
    }
    t /= test;
    // std::cout << mode << ' ' << block_size_x << ' ' << block_size_y << ' ' << "Exec-time: " << t << " ms" << std::endl;
    if (! use_shared_memory)
        oFile << "| " << block_size_x << " × " << block_size_y << " | " << t << " ms | ";
    else
        oFile << t << " ms |" << std::endl;
}

int main()
{
    oFile.open("result.md", std::ios::out);
    oFile << "| block size | 不使用 shared memory 时间 | 使用 shared memory 时间 |\n| ---------- | -------------------- | ------------------- |\n";
    prepareInput();
    for (int i = 32; i <= 1024; i += 32) // blockDim.x
        for (int j = 1; j <= 32; ++j)    // blockDim.y
        {
            if (i * j > 1024)
                break;
            test_func(i, j, false, 0); // default: no validation
            test_func(i, j, true, 0);
        }
    oFile.close();
    return 0;
}
